#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include "decimal.h"
#include "extcolordefs.h"
#include "text.h"

#ifndef _USE_MATH_DEFINES
#define _USE_MATH_DEFINES
#endif

#include <iostream>
#include <math.h>

// Optimal values (calculated)

#define CUDA_BLOCK_SIDE_LENGTH 32
#define CUDA_GAUSSIAN_BLUR_BLOCK_SIDE_LENGTH 16
#define CUDA_THREADS_PER_BLOCK (CUDA_BLOCK_SIDE_LENGTH*CUDA_BLOCK_SIDE_LENGTH)
#define CUDA_3_CHANNEL_MAX_FILTER_SIZE 19
#define CUDA_1_CHANNEL_MAX_FILTER_SIZE 19
#define CUDA_CANNY_LOW_COLOR 0xff000000
#define CUDA_CANNY_HIGH_COLOR 0xffffffff

#if 0==1
#define __shared__
#define __global__
#define __device__
#define __host__
#endif

void cudaLog(const char *str)
{
    std::cout<<str<<std::endl;
}

__global__ void device_getBWImage(uint32_t *imageData,int width,uint32_t *bwImageDataOut)
{
    int threadId=threadIdx.x;
    int y=blockIdx.y;
    int blockIdInLine=blockIdx.x;
    int x=blockIdInLine*blockDim.x+threadId;

    if(x>=width)
        return;

    size_t pos=y*width+x;
    uint32_t color=imageData[pos];
    float r=getFRed(color);
    float g=getFGreen(color);
    float b=getFBlue(color);

    uint8_t component=round((0.2126f*r+0.7152f*g+0.0722f*b)*255.0f);

    bwImageDataOut[pos]=component;
}

__global__ void device_cannyEdgeDetect_stage1(uint32_t *imageData,int width,int height,int filterSize,float deviation,float *gaussianBwArrayOut)
{
    const int blockImageSectionSingleChannelSizeSqrt=CUDA_1_CHANNEL_MAX_FILTER_SIZE*2+CUDA_BLOCK_SIDE_LENGTH;
    __shared__ float blockImageSection[blockImageSectionSingleChannelSizeSqrt*blockImageSectionSingleChannelSizeSqrt];

    int filterSizeInPixels=2*filterSize+1;
    int blockHorId=blockIdx.x;
    int blockVerId=blockIdx.y;
    int relativeX=threadIdx.x;
    int x=blockHorId*blockDim.x+relativeX;
    int relativeY=threadIdx.y;
    int y=blockVerId*blockDim.y+relativeY;

    int rowSize=2*filterSize+blockDim.x;
    int columnSize=2*filterSize+blockDim.y;
    int topOffsetRows=filterSize;

    if(x>=width||y>=height)
        return;

    size_t pos=y*width+x;
    uint32_t color=imageData[pos];
    // Use getFBWComponentFromColor
    float fbwComponent=getFBWComponentFromColor(color);

    // Copy pixels

    // Copy this pixel from source image

    int posInBlockImageSectionData=(topOffsetRows+relativeY)*rowSize+filterSize+relativeX;
    blockImageSection[posInBlockImageSectionData]=fbwComponent;

    // Copy extra pixels, if needed

    int effectiveBlockWidth=blockHorId==gridDim.x-1?(width-blockHorId*blockDim.x):blockDim.x;
    int effectiveBlockHeight=blockVerId==gridDim.y-1?(height-blockVerId*blockDim.y):blockDim.y;

    int relativeLeftExtra=-filterSize+relativeX;
    int relativeRightExtra=relativeX+filterSize;
    int relativeTopExtra=-filterSize+relativeY;
    int relativeBottomExtra=relativeY+filterSize;

    bool copyExtraFromLeft=false;
    bool leftOverflow=false;
    bool copyExtraFromRight=false;
    bool rightOverflow=false;
    int absLeftExtra;
    int absRightExtra;
    uint32_t extraColor;

    if(copyExtraFromLeft=(relativeLeftExtra<0))
    {
        absLeftExtra=x-filterSize;
        if(leftOverflow=(absLeftExtra<0))
        {
            extraColor=imageData[y*width];
            blockImageSection[posInBlockImageSectionData-filterSize]=getFBWComponentFromColor(extraColor);
        }
        else
        {
            extraColor=imageData[y*width+absLeftExtra];
            blockImageSection[posInBlockImageSectionData-filterSize]=getFBWComponentFromColor(extraColor);
        }
    }
    if(copyExtraFromRight=(relativeRightExtra>=effectiveBlockWidth))
    {
        absRightExtra=x+filterSize;
        if(rightOverflow=(absRightExtra>=width))
        {
            extraColor=imageData[y*width+width-1];
            blockImageSection[posInBlockImageSectionData+filterSize]=getFBWComponentFromColor(extraColor);
        }
        else
        {
            extraColor=imageData[y*width+absRightExtra];
            blockImageSection[posInBlockImageSectionData+filterSize]=getFBWComponentFromColor(extraColor);
        }
    }
    if(relativeTopExtra<0)
    {
        int absTopExtra=y-filterSize;
        if(absTopExtra<0)
        {
            extraColor=imageData[(-absTopExtra)*width+x];
            blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)]=getFBWComponentFromColor(extraColor);
            if(copyExtraFromLeft)
            {
                if(leftOverflow)
                {
                    extraColor=imageData[(-absTopExtra)*width];
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)-filterSize]=getFBWComponentFromColor(extraColor);
                }
                else
                {
                    extraColor=imageData[(-absTopExtra)*width+absLeftExtra];
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)-filterSize]=getFBWComponentFromColor(extraColor);
                }
            }
            if(copyExtraFromRight)
            {
                if(rightOverflow)
                {
                    extraColor=imageData[(-absTopExtra)*width+width-1];
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)+filterSize]=getFBWComponentFromColor(extraColor);
                }
                else
                {
                    extraColor=imageData[(-absTopExtra)*width+absRightExtra];
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)+filterSize]=getFBWComponentFromColor(extraColor);
                }
            }
        }
        else
        {
            extraColor=imageData[absTopExtra*width+x];
            blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)]=getFBWComponentFromColor(extraColor);
            if(copyExtraFromLeft)
            {
                if(leftOverflow)
                {
                    extraColor=imageData[absTopExtra*width];
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)-filterSize]=getFBWComponentFromColor(extraColor);
                }
                else
                {
                    extraColor=imageData[absTopExtra*width+absLeftExtra];
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)-filterSize]=getFBWComponentFromColor(extraColor);
                }
            }
            if(copyExtraFromRight)
            {
                if(rightOverflow)
                {
                    extraColor=imageData[absTopExtra*width+width-1];
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)+filterSize]=getFBWComponentFromColor(extraColor);
                }
                else
                {
                    extraColor=imageData[absTopExtra*width+absRightExtra];
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)+filterSize]=getFBWComponentFromColor(extraColor);
                }
            }
        }
    }
    if(relativeBottomExtra>=effectiveBlockHeight)
    {
        int absBottomExtra=y+filterSize;
        if(absBottomExtra>=height)
        {
            extraColor=imageData[(height-(absBottomExtra-height+1))*width+x];
            blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)]=getFBWComponentFromColor(extraColor);
            if(copyExtraFromLeft)
            {
                if(leftOverflow)
                {
                    extraColor=imageData[(height-(absBottomExtra-height+1))*width];
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)-filterSize]=getFBWComponentFromColor(extraColor);
                }
                else
                {
                    extraColor=imageData[(height-(absBottomExtra-height+1))*width+absLeftExtra];
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)-filterSize]=getFBWComponentFromColor(extraColor);
                }
            }
            if(copyExtraFromRight)
            {
                if(rightOverflow)
                {
                    extraColor=imageData[(height-(absBottomExtra-height+1))*width+width-1];
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)+filterSize]=getFBWComponentFromColor(extraColor);
                }
                else
                {
                    extraColor=imageData[(height-(absBottomExtra-height+1))*width+absRightExtra];
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)+filterSize]=getFBWComponentFromColor(extraColor);
                }
            }
        }
        else
        {
            extraColor=imageData[absBottomExtra*width+x];
            blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)]=getFBWComponentFromColor(extraColor);
            if(copyExtraFromLeft)
            {
                if(leftOverflow)
                {
                    extraColor=imageData[absBottomExtra*width];
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)-filterSize]=getFBWComponentFromColor(extraColor);
                }
                else
                {
                    extraColor=imageData[absBottomExtra*width+absLeftExtra];
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)-filterSize]=getFBWComponentFromColor(extraColor);
                }
            }
            if(copyExtraFromRight)
            {
                if(rightOverflow)
                {
                    extraColor=imageData[absBottomExtra*width+width-1];
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)+filterSize]=getFBWComponentFromColor(extraColor);
                }
                else
                {
                    extraColor=imageData[absBottomExtra*width+absRightExtra];
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)+filterSize]=getFBWComponentFromColor(extraColor);
                }
            }
        }
    }

    float preFactor=(1.0f/(2.0f*M_PI_F*deviation*deviation));
    float preFactor2=(2.0f*deviation*deviation);
    float filterFactors[CUDA_3_CHANNEL_MAX_FILTER_SIZE*CUDA_3_CHANNEL_MAX_FILTER_SIZE];
    for(int filterY=0;filterY<filterSizeInPixels;filterY++)
    {
        int offset=filterY*filterSizeInPixels;
        for(int filterX=0;filterX<filterSizeInPixels;filterX++)
        {
            int n1=filterX+1-filterSize-1;
            int n2=filterY+1-filterSize-1;
            float factor=preFactor*exp(-(((float)(n1*n1+n2*n2))/preFactor2));
            filterFactors[offset+filterX]=factor;
        }
    }

    __syncthreads();

    int relativeXWithBorderPixels=relativeX+filterSize;
    int relativeYWithBorderPixels=relativeY+filterSize;
    float pixelValueSum=0.0f;

    for(int yOfFilter=0;yOfFilter<filterSizeInPixels;yOfFilter++)
    {
        int yWithFilter=-filterSize+relativeYWithBorderPixels+yOfFilter;
         // Use symmetry to compensate for missing pixels (in order to avoid dark borders)
        if(yWithFilter<0)
            yWithFilter=(relativeYWithBorderPixels+filterSize)-yOfFilter;
        else if(yWithFilter>=columnSize)
            yWithFilter=(relativeYWithBorderPixels-filterSize)+(filterSizeInPixels-1-yOfFilter);
        for(int xOfFilter=0;xOfFilter<filterSizeInPixels;xOfFilter++)
        {
            int xWithFilter=-filterSize+relativeXWithBorderPixels+xOfFilter;
            // Use symmetry to compensate for missing pixels (in order to avoid dark borders)
            if(xWithFilter<0)
                xWithFilter=(relativeXWithBorderPixels+filterSize)-xOfFilter;
            else if(xWithFilter>=rowSize)
                xWithFilter=(relativeXWithBorderPixels-filterSize)+(filterSizeInPixels-1-xOfFilter);

            float factor=filterFactors[yOfFilter*filterSizeInPixels+xOfFilter];
            pixelValueSum+=blockImageSection[yWithFilter*rowSize+xWithFilter]*factor;
        }
    }

    gaussianBwArrayOut[pos]=pixelValueSum;
}

__global__ void device_cannyEdgeDetect_stage2(int width,int height,int filterSize,float *gaussianBwArray,float *gradientArrayOut,float *gradientAtan2ArrayOut)
{
    __shared__ float blockBWCache[(CUDA_BLOCK_SIDE_LENGTH+2)*(CUDA_BLOCK_SIDE_LENGTH+2)];

    int blockHorId=blockIdx.x;
    int blockVerId=blockIdx.y;
    int relativeX=threadIdx.x;
    int x=blockHorId*blockDim.x+relativeX;
    int relativeY=threadIdx.y;
    int y=blockVerId*blockDim.y+relativeY;

    if(x>=width||y>=height)
        return;

    size_t pos=y*width+x;

    float bw=gaussianBwArray[pos];

    int blockBWCacheRowSize=blockDim.x+2;

    int blockBWCachePos=(1+relativeY)*blockBWCacheRowSize+1+relativeX;
    blockBWCache[blockBWCachePos]=bw;

    bool hasTop=(y-1)>=0;
    bool hasBottom=(y+1)<height;
    bool hasLeft=(x-1)>=0;
    bool hasRight=(x+1)<width;

    bool leftmostInBlock=relativeX==0;
    bool rightmostInBlock=relativeX==blockDim.x-1||!(hasRight); // Blocks on the borders of the image have other border pixels
    bool topmostInBlock=relativeY==0;
    bool bottommostInBlock=relativeY==blockDim.y-1||!(hasBottom); // Blocks on the borders of the image have other border pixels

    if(leftmostInBlock&&hasLeft)
    {
        blockBWCache[blockBWCachePos-1]=gaussianBwArray[pos-1];
    }
    else if(rightmostInBlock&&hasRight)
    {
        blockBWCache[blockBWCachePos+1]=gaussianBwArray[pos+1];
    }

    if(topmostInBlock&&hasTop)
    {
        blockBWCache[blockBWCachePos-blockBWCacheRowSize]=gaussianBwArray[pos-width];
        if(leftmostInBlock&&hasLeft)
        {
            blockBWCache[blockBWCachePos-blockBWCacheRowSize-1]=gaussianBwArray[pos-width-1];
        }
        else if(rightmostInBlock&&hasRight)
        {
            blockBWCache[blockBWCachePos-blockBWCacheRowSize+1]=gaussianBwArray[pos-width+1];
        }
    }
    else if(bottommostInBlock&&hasBottom)
    {
        blockBWCache[blockBWCachePos+blockBWCacheRowSize]=gaussianBwArray[pos+width];
        if(leftmostInBlock&&hasLeft)
        {
            blockBWCache[blockBWCachePos+blockBWCacheRowSize-1]=gaussianBwArray[pos+width-1];
        }
        else if(rightmostInBlock&&hasRight)
        {
            blockBWCache[blockBWCachePos+blockBWCacheRowSize+1]=gaussianBwArray[pos+width+1];
        }
    }

    float accXValueSum=0.0f;
    float accYValueSum=0.0f;

    int relativeLeftX=relativeX-1;
    int relativeRightX=relativeX+1;
    int relativeYWithOffset=(1+relativeY)*blockBWCacheRowSize+1;
    int relativeTopYWithOffset=(1+relativeY-1)*blockBWCacheRowSize+1;
    int relativeBottomYWithOffset=(1+relativeY+1)*blockBWCacheRowSize+1;

    __syncthreads(); // Make sure blockBWCache is filled out correctly (see above)

    if(hasLeft)
    {
        if(hasTop)
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeTopYWithOffset+relativeLeftX]*(-1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeTopYWithOffset+relativeLeftX]*(-1.0f);
        }
        else
        {
            // Extend the image by 1 pixel on each side and using the outmost pixels in order to
            // avoid false positives on the borders of the image

            // X accumulator
            accXValueSum+=blockBWCache[relativeYWithOffset+relativeLeftX]*(-1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeYWithOffset+relativeLeftX]*(-1.0f);
        }
        if(hasBottom)
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeBottomYWithOffset+relativeLeftX]*(-1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeBottomYWithOffset+relativeLeftX]*(1.0f);
        }
        else
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeYWithOffset+relativeLeftX]*(-1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeYWithOffset+relativeLeftX]*(1.0f);
        }
        // X accumulator
        accXValueSum+=blockBWCache[relativeYWithOffset+relativeLeftX]*(-2.0f);
    }
    else
    {
        if(hasTop)
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeTopYWithOffset+relativeX]*(-1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeTopYWithOffset+relativeX]*(-1.0f);
        }
        else
        {
            // Extend the image by 1 pixel on each side and using the outmost pixels in order to
            // avoid false positives on the borders of the image

            // X accumulator
            accXValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(-1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(-1.0f);
        }
        if(hasBottom)
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeBottomYWithOffset+relativeX]*(-1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeBottomYWithOffset+relativeX]*(1.0f);
        }
        else
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(-1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(1.0f);
        }
        // X accumulator
        accXValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(-2.0f);
    }
    if(hasRight)
    {
        if(hasTop)
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeTopYWithOffset+relativeRightX]*(1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeTopYWithOffset+relativeRightX]*(-1.0f);
        }
        else
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeYWithOffset+relativeRightX]*(1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeYWithOffset+relativeRightX]*(-1.0f);
        }
        if(hasBottom)
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeBottomYWithOffset+relativeRightX]*(1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeBottomYWithOffset+relativeRightX]*(1.0f);
        }
        else
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeYWithOffset+relativeRightX]*(1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeYWithOffset+relativeRightX]*(1.0f);
        }
        // X accumulator
        accXValueSum+=blockBWCache[relativeYWithOffset+relativeRightX]*(2.0f);
    }
    else
    {
        if(hasTop)
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeTopYWithOffset+relativeX]*(1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeTopYWithOffset+relativeX]*(-1.0f);
        }
        else
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(-1.0f);
        }
        if(hasBottom)
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeBottomYWithOffset+relativeX]*(1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeBottomYWithOffset+relativeX]*(1.0f);
        }
        else
        {
            // X accumulator
            accXValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(1.0f);
            // Y accumulator
            accYValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(1.0f);
        }
        // X accumulator
        accXValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(2.0f);
    }
    if(hasTop)
    {
        // Y accumulator
        accYValueSum+=blockBWCache[relativeTopYWithOffset+relativeX]*(-2.0f);
    }
    else
    {
        // Y accumulator
        accYValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(-2.0f);
    }
    if(hasBottom)
    {
        // Y accumulator
        accYValueSum+=blockBWCache[relativeBottomYWithOffset+relativeX]*(2.0f);
    }
    else
    {
        // Y accumulator
        accYValueSum+=blockBWCache[relativeYWithOffset+relativeX]*(2.0f);
    }
    // The pixel in the center has 0.0f as its factor for both filters.

    // Result
    float gradient=sqrt(accXValueSum*accXValueSum+accYValueSum*accYValueSum);
    float gradientAtan2=atan2(accXValueSum,accYValueSum);

    gradientArrayOut[pos]=gradient;
    gradientAtan2ArrayOut[pos]=gradientAtan2;
}

__global__ void device_cannyEdgeDetect_stage3(int width,int height,float highTreshold,float lowTreshold,float *gradientArray,float *gradientAtan2Array,uint8_t *preResultArrayOut)
{
    __shared__ float blockGradientCache[(CUDA_BLOCK_SIDE_LENGTH+2)*(CUDA_BLOCK_SIDE_LENGTH+2)];

    int blockHorId=blockIdx.x;
    int blockVerId=blockIdx.y;
    int relativeX=threadIdx.x;
    int x=blockHorId*blockDim.x+relativeX;
    int relativeY=threadIdx.y;
    int y=blockVerId*blockDim.y+relativeY;

    if(x>=width||y>=height)
        return;

    size_t pos=y*width+x;

    float gradient=gradientArray[pos];

    int blockGradientCacheRowSize=blockDim.x+2;

    int blockGradientCachePos=(1+relativeY)*blockGradientCacheRowSize+1+relativeX;
    blockGradientCache[blockGradientCachePos]=gradient;

    bool hasTop=(y-1)>=0;
    bool hasBottom=(y+1)<height;
    bool hasLeft=(x-1)>=0;
    bool hasRight=(x+1)<width;

    bool leftmostInBlock=relativeX==0;
    bool rightmostInBlock=relativeX==blockDim.x-1||!(hasRight); // Blocks on the borders of the image have other border pixels
    bool topmostInBlock=relativeY==0;
    bool bottommostInBlock=relativeY==blockDim.y-1||!(hasBottom); // Blocks on the borders of the image have other border pixels

    if(leftmostInBlock&&hasLeft)
    {
        blockGradientCache[blockGradientCachePos-1]=gradientArray[pos-1];
    }
    else if(rightmostInBlock&&hasRight)
    {
        blockGradientCache[blockGradientCachePos+1]=gradientArray[pos+1];
    }

    if(topmostInBlock&&hasTop)
    {
        blockGradientCache[blockGradientCachePos-blockGradientCacheRowSize]=gradientArray[pos-width];
        if(leftmostInBlock&&hasLeft)
        {
            blockGradientCache[blockGradientCachePos-blockGradientCacheRowSize-1]=gradientArray[pos-width-1];
        }
        else if(rightmostInBlock&&hasRight)
        {
            blockGradientCache[blockGradientCachePos-blockGradientCacheRowSize+1]=gradientArray[pos-width+1];
        }
    }
    else if(bottommostInBlock&&hasBottom)
    {
        blockGradientCache[blockGradientCachePos+blockGradientCacheRowSize]=gradientArray[pos+width];
        if(leftmostInBlock&&hasLeft)
        {
            blockGradientCache[blockGradientCachePos+blockGradientCacheRowSize-1]=gradientArray[pos+width-1];
        }
        else if(rightmostInBlock&&hasRight)
        {
            blockGradientCache[blockGradientCachePos+blockGradientCacheRowSize+1]=gradientArray[pos+width+1];
        }
    }

    __syncthreads();

    float angle=gradientAtan2Array[pos];

    int rAngle=(int)round(angle/(0.25f*M_PI_F)); // 45 degrees
    if(rAngle<0)
        rAngle=4+rAngle;
    bool eastWest=rAngle==0||rAngle==4; // The first section is split (one half on each end)
    bool northEastSouthWest=rAngle==1;
    bool northSouth=rAngle==2;
    bool northWestSouthEast=rAngle==3;

    float preResult;

    if(eastWest)
    {
        float neighborPixelValue1=hasTop?blockGradientCache[blockGradientCachePos-blockGradientCacheRowSize]:gradient;
        float neighborPixelValue2=hasBottom?blockGradientCache[blockGradientCachePos+blockGradientCacheRowSize]:gradient;
        if(gradient>neighborPixelValue1&&gradient>neighborPixelValue2)
            preResult=gradient;
        else
            preResult=0.0f;
    }
    else if(northEastSouthWest)
    {
        float neighborPixelValue1=hasLeft&&hasTop?blockGradientCache[blockGradientCachePos-blockGradientCacheRowSize-1]:(hasLeft?blockGradientCache[blockGradientCachePos-1]:(hasTop?blockGradientCache[blockGradientCachePos-blockGradientCacheRowSize]:gradient));
        float neighborPixelValue2=hasRight&&hasBottom?blockGradientCache[blockGradientCachePos+blockGradientCacheRowSize+1]:(hasRight?blockGradientCache[blockGradientCachePos+1]:(hasBottom?blockGradientCache[blockGradientCachePos+blockGradientCacheRowSize]:gradient));
        if(gradient>neighborPixelValue1&&gradient>neighborPixelValue2)
            preResult=gradient;
        else
            preResult=0.0f;
    }
    else if(northSouth)
    {
        float neighborPixelValue1=hasLeft?blockGradientCache[blockGradientCachePos-1]:gradient;
        float neighborPixelValue2=hasRight?blockGradientCache[blockGradientCachePos+1]:gradient;
        if(gradient>neighborPixelValue1&&gradient>neighborPixelValue2)
            preResult=gradient;
        else
            preResult=0.0f;
    }
    else if(northWestSouthEast)
    {
        float neighborPixelValue1=hasRight&&hasTop?blockGradientCache[blockGradientCachePos-blockGradientCacheRowSize+1]:(hasRight?blockGradientCache[blockGradientCachePos+1]:(hasTop?blockGradientCache[blockGradientCachePos-blockGradientCacheRowSize]:gradient));
        float neighborPixelValue2=hasLeft&&hasBottom?blockGradientCache[blockGradientCachePos+blockGradientCacheRowSize-1]:(hasLeft?blockGradientCache[blockGradientCachePos-1]:(hasBottom?blockGradientCache[blockGradientCachePos+blockGradientCacheRowSize]:gradient));
        if(gradient>neighborPixelValue1&&gradient>neighborPixelValue2)
            preResult=gradient;
        else
            preResult=0.0f;
    }

    uint8_t preResultOut;

    if(preResult<lowTreshold)
        preResultOut=0;
    else if(preResult<highTreshold)
        preResultOut=1;
    else // if(preResult>=highTreshold)
        preResultOut=2;

    preResultArrayOut[pos]=preResultOut;
}

__global__ void device_cannyEdgeDetect_stage4(int width,int height,uint8_t *preResultArray,uint32_t *imageDataOut)
{
    __shared__ uint8_t blockPreResultCache[(CUDA_BLOCK_SIDE_LENGTH+2)*(CUDA_BLOCK_SIDE_LENGTH+2)];

    int blockHorId=blockIdx.x;
    int blockVerId=blockIdx.y;
    int relativeX=threadIdx.x;
    int x=blockHorId*blockDim.x+relativeX;
    int relativeY=threadIdx.y;
    int y=blockVerId*blockDim.y+relativeY;

    if(x>=width||y>=height)
        return;

    size_t pos=y*width+x;

    uint8_t preResult=preResultArray[pos];

    int blockPreResultCacheRowSize=blockDim.x+2;

    int blockPreResultCachePos=(1+relativeY)*blockPreResultCacheRowSize+1+relativeX;
    blockPreResultCache[blockPreResultCachePos]=preResult;

    bool hasTop=(y-1)>=0;
    bool hasBottom=(y+1)<height;
    bool hasLeft=(x-1)>=0;
    bool hasRight=(x+1)<width;

    bool leftmostInBlock=relativeX==0;
    bool rightmostInBlock=relativeX==blockDim.x-1||!(hasRight); // Blocks on the borders of the image have other border pixels
    bool topmostInBlock=relativeY==0;
    bool bottommostInBlock=relativeY==blockDim.y-1||!(hasBottom); // Blocks on the borders of the image have other border pixels

    if(leftmostInBlock&&hasLeft)
    {
        blockPreResultCache[blockPreResultCachePos-1]=preResultArray[pos-1];
    }
    else if(rightmostInBlock&&hasRight)
    {
        blockPreResultCache[blockPreResultCachePos+1]=preResultArray[pos+1];
    }

    if(topmostInBlock&&hasTop)
    {
        blockPreResultCache[blockPreResultCachePos-blockPreResultCacheRowSize]=preResultArray[pos-width];
        if(leftmostInBlock&&hasLeft)
        {
            blockPreResultCache[blockPreResultCachePos-blockPreResultCacheRowSize-1]=preResultArray[pos-width-1];
        }
        else if(rightmostInBlock&&hasRight)
        {
            blockPreResultCache[blockPreResultCachePos-blockPreResultCacheRowSize+1]=preResultArray[pos-width+1];
        }
    }
    else if(bottommostInBlock&&hasBottom)
    {
        blockPreResultCache[blockPreResultCachePos+blockPreResultCacheRowSize]=preResultArray[pos+width];
        if(leftmostInBlock&&hasLeft)
        {
            blockPreResultCache[blockPreResultCachePos+blockPreResultCacheRowSize-1]=preResultArray[pos+width-1];
        }
        else if(rightmostInBlock&&hasRight)
        {
            blockPreResultCache[blockPreResultCachePos+blockPreResultCacheRowSize+1]=preResultArray[pos+width+1];
        }
    }

    __syncthreads();

    if(preResult==0)
        imageDataOut[pos]=CUDA_CANNY_LOW_COLOR;
    else if(preResult==2)
        imageDataOut[pos]=CUDA_CANNY_HIGH_COLOR;
    else // Weak edge; decide whether to keep it (if at least one neighboring pixel is a strong edge)
    {
        if(hasLeft)
        {
            if(blockPreResultCache[blockPreResultCachePos-1]==2) // preResult[y*width+leftX]==1.0f
            {
                imageDataOut[pos]=CUDA_CANNY_HIGH_COLOR;
                return;
            }
            if(hasTop)
            {
                if(blockPreResultCache[blockPreResultCachePos-blockPreResultCacheRowSize]==2) // preResult[topY*width+leftX]==1.0f
                {
                    imageDataOut[pos]=CUDA_CANNY_HIGH_COLOR;
                    return;
                }
            }
            if(hasBottom)
            {
                if(blockPreResultCache[blockPreResultCachePos+blockPreResultCacheRowSize]==2) // preResult[bottomY*width+leftX]==1.0f
                {
                    imageDataOut[pos]=CUDA_CANNY_HIGH_COLOR;
                    return;
                }
            }
        }
        if(hasRight)
        {
            if(blockPreResultCache[blockPreResultCachePos+1]==2) // preResult[y*width+rightX]==1.0f
            {
                imageDataOut[pos]=CUDA_CANNY_HIGH_COLOR;
                return;
            }
            if(hasTop)
            {
                if(blockPreResultCache[blockPreResultCachePos-blockPreResultCacheRowSize+1]==2) // preResult[topY*width+rightX]==1.0f
                {
                    imageDataOut[pos]=CUDA_CANNY_HIGH_COLOR;
                    return;
                }
            }
            if(hasBottom)
            {
                if(blockPreResultCache[blockPreResultCachePos+blockPreResultCacheRowSize+1]==2) // preResult[bottomY*width+rightX]==1.0f
                {
                    imageDataOut[pos]=CUDA_CANNY_HIGH_COLOR;
                    return;
                }
            }
        }
        if(hasTop)
        {
            if(blockPreResultCache[blockPreResultCachePos-blockPreResultCacheRowSize]==2) // preResult[topY*width+x]==1.0f
            {
                imageDataOut[pos]=CUDA_CANNY_HIGH_COLOR;
                return;
            }
        }
        if(hasBottom)
        {
            if(blockPreResultCache[blockPreResultCachePos+blockPreResultCacheRowSize]==2) // preResult[bottomY*width+x]==1.0f
            {
                imageDataOut[pos]=CUDA_CANNY_HIGH_COLOR;
                return;
            }
        }
        imageDataOut[pos]=CUDA_CANNY_LOW_COLOR;
    }
}

__global__ void device_sobelEdgeDetect(uint32_t *imageData,int width,int height,float amplifier,uint32_t *imageDataOut)
{
    __shared__ float blockImageSection[(2+CUDA_BLOCK_SIDE_LENGTH)*(2+CUDA_BLOCK_SIDE_LENGTH)]; // Left/right/top/bottom pixel rows included

    int blockHorId=blockIdx.x;
    int blockVerId=blockIdx.y;
    int relativeX=threadIdx.x;
    int x=blockHorId*blockDim.x+relativeX;
    int relativeY=threadIdx.y;
    int y=blockVerId*blockDim.y+relativeY;

    if(x>=width||y>=height)
        return;

    size_t pos=y*width+x;
    uint32_t color=imageData[pos];

    // Fill up blockImageSection with B/W versions of pixels

    // This pixel:
    size_t blockImageSectionPos=(1+relativeY)*(CUDA_BLOCK_SIDE_LENGTH+2)+(1+relativeX); // "1+...": border pixels
    float component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
    blockImageSection[blockImageSectionPos]=component;

    bool leftmostInBlock=false;
    bool rightmostInBlock=false;
    bool topmostInBlock=false;
    bool bottommostInBlock=false;

    bool leftmostInImage;
    bool topmostInImage;
    bool rightmostInImage=x==width-1;
    bool bottommostInImage=y==height-1;

    // Extend the image by 1 pixel on each side and using the outmost pixels in order to
    // avoid false positives on the borders of the image

    if(rightmostInBlock=(relativeX==CUDA_BLOCK_SIDE_LENGTH-1||rightmostInImage))
    {
        if(rightmostInImage)
        {
            color=imageData[pos];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos+1]=component;
        }
        else
        {
            color=imageData[pos+1];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos+1]=component;
        }
    }
    else if(leftmostInBlock=(relativeX==0))
    {
        if(leftmostInImage=(x==0))
        {
            color=imageData[pos];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos-1]=component;
        }
        else
        {
            color=imageData[pos-1];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos-1]=component;
        }
    }

    if(bottommostInBlock=(relativeY==CUDA_BLOCK_SIDE_LENGTH-1||bottommostInImage))
    {
        if(bottommostInImage)
        {
            color=imageData[pos];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+2)]=component;
            if(leftmostInBlock)
            {
                if(leftmostInImage)
                {
                    //color=imageData[pos]; // Already set
                    //component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+2)-1]=component;
                }
                else
                {
                    color=imageData[pos-1];
                    component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+2)-1]=component;
                }
            }
            else if(rightmostInBlock)
            {
                if(rightmostInImage)
                {
                    //color=imageData[pos]; // Already set
                    //component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+2)+1]=component;
                }
                else
                {
                    color=imageData[pos+1];
                    component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+2)+1]=component;
                }
            }
        }
        else
        {
            color=imageData[pos+width];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+2)]=component;
            if(leftmostInBlock)
            {
                if(leftmostInImage)
                {
                    //color=imageData[pos+width]; // Already set
                    //component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+2)-1]=component;
                }
                else
                {
                    color=imageData[pos+width-1];
                    component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+2)-1]=component;
                }
            }
            else if(rightmostInBlock)
            {
                if(rightmostInImage)
                {
                    //color=imageData[pos+width]; // Already set
                    //component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+2)+1]=component;
                }
                else
                {
                    color=imageData[pos+width+1];
                    component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+2)+1]=component;
                }
            }
        }
    }
    else if(topmostInBlock=(relativeY==0))
    {
        if(topmostInImage=(y==0))
        {
            color=imageData[pos];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos-(CUDA_BLOCK_SIDE_LENGTH+2)]=component;
            if(leftmostInBlock)
            {
                if(leftmostInImage)
                {
                    //color=imageData[pos]; // Already set
                    //component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos-(CUDA_BLOCK_SIDE_LENGTH+2)-1]=component;
                }
                else
                {
                    color=imageData[pos-1];
                    component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos-(CUDA_BLOCK_SIDE_LENGTH+2)-1]=component;
                }
            }
            else if(rightmostInBlock)
            {
                if(rightmostInImage)
                {
                    //color=imageData[pos]; // Already set
                    //component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos-(CUDA_BLOCK_SIDE_LENGTH+2)+1]=component;
                }
                else
                {
                    color=imageData[pos+1];
                    component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos-(CUDA_BLOCK_SIDE_LENGTH+2)+1]=component;
                }
            }
        }
        else
        {
            color=imageData[pos-width];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos-(CUDA_BLOCK_SIDE_LENGTH+2)]=component;
            if(leftmostInBlock)
            {
                if(leftmostInImage)
                {
                    //color=imageData[pos-width]; // Already set
                    //component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos-(CUDA_BLOCK_SIDE_LENGTH+2)-1]=component;
                }
                else
                {
                    color=imageData[pos-width-1];
                    component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos-(CUDA_BLOCK_SIDE_LENGTH+2)-1]=component;
                }
            }
            else if(rightmostInBlock)
            {
                if(rightmostInImage)
                {
                    //color=imageData[pos-width]; // Already set
                    //component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos-(CUDA_BLOCK_SIDE_LENGTH+2)+1]=component;
                }
                else
                {
                    color=imageData[pos-width+1];
                    component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos-(CUDA_BLOCK_SIDE_LENGTH+2)+1]=component;
                }
            }
        }
    }

    __syncthreads();

    float accXValueSum=0.0f;
    float accYValueSum=0.0f;

    // Left/top

    // X accumulator
    accXValueSum+=blockImageSection[(1+relativeY-1)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX-1]*(-1.0f);
    // Y accumulator
    accYValueSum+=blockImageSection[(1+relativeY-1)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX-1]*(-1.0f);

    // Left/bottom

    // X accumulator
    accXValueSum+=blockImageSection[(1+relativeY+1)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX-1]*(-1.0f);
    // Y accumulator
    accYValueSum+=blockImageSection[(1+relativeY+1)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX-1]*(1.0f);

    // Left

    // X accumulator
    accXValueSum+=blockImageSection[(1+relativeY)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX-1]*(-2.0f);

    // Right/top

    // X accumulator
    accXValueSum+=blockImageSection[(1+relativeY-1)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX+1]*(1.0f);
    // Y accumulator
    accYValueSum+=blockImageSection[(1+relativeY-1)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX+1]*(-1.0f);

    // Right/bottom

    // X accumulator
    accXValueSum+=blockImageSection[(1+relativeY+1)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX+1]*(1.0f);
    // Y accumulator
    accYValueSum+=blockImageSection[(1+relativeY+1)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX+1]*(1.0f);

    // Right

    // X accumulator
    accXValueSum+=blockImageSection[(1+relativeY)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX+1]*(2.0f);

    // Top

    // Y accumulator
    accYValueSum+=blockImageSection[(1+relativeY-1)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX]*(-2.0f);

    // Bottom

    // Y accumulator
    accYValueSum+=blockImageSection[(1+relativeY+1)*(CUDA_BLOCK_SIDE_LENGTH+2)+1+relativeX]*(2.0f);

    // Final result

    component=amplifier*sqrt(accXValueSum*accXValueSum+accYValueSum*accYValueSum);
    uint8_t componentOut=(uint8_t)round(component*255.0);
    imageDataOut[pos]=getColor(255,componentOut,componentOut,componentOut);
}

__global__ void device_robertsEdgeDetect(uint32_t *imageData,int width,int height,float amplifier,uint32_t *imageDataOut)
{
    __shared__ float blockImageSection[(1+CUDA_BLOCK_SIDE_LENGTH)*(1+CUDA_BLOCK_SIDE_LENGTH)]; // Right/bottom pixel rows included

    int blockHorId=blockIdx.x;
    int blockVerId=blockIdx.y;
    int relativeX=threadIdx.x;
    int x=blockHorId*blockDim.x+relativeX;
    int relativeY=threadIdx.y;
    int y=blockVerId*blockDim.y+relativeY;

    if(x>=width||y>=height)
        return;

    int pos=y*width+x;
    uint32_t color=imageData[pos];

    // Fill up blockImageSection with B/W versions of pixels

    // Exact same pixel:
    size_t blockImageSectionPos=relativeY*(CUDA_BLOCK_SIDE_LENGTH+1)+relativeX; // "1+...": border pixels
    float component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
    blockImageSection[blockImageSectionPos]=component;

    bool rightmostInBlock=false;
    bool bottommostInBlock=false;

    bool rightmostInImage=x==width-1;
    bool bottommostInImage=y==height-1;

    // Extend the image by 1 pixel on each side and using the outmost pixels in order to
    // avoid false positives on the borders of the image

    if(rightmostInBlock=(relativeX==CUDA_BLOCK_SIDE_LENGTH-1||rightmostInImage))
    {
        if(rightmostInImage)
        {
            color=imageData[pos];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos+1]=component;
        }
        else
        {
            color=imageData[pos+1];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos+1]=component;
        }
    }

    if(bottommostInBlock=(relativeY==CUDA_BLOCK_SIDE_LENGTH-1||bottommostInImage))
    {
        if(bottommostInImage)
        {
            color=imageData[pos];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+1)]=component;
            if(rightmostInBlock)
            {
                if(rightmostInImage)
                {
                    //color=imageData[pos]; // Already set
                    //component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+1)+1]=component;
                }
                else
                {
                    color=imageData[pos+1];
                    component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+1)+1]=component;
                }
            }
        }
        else
        {
            color=imageData[pos+width];
            component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
            blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+1)]=component;
            if(rightmostInBlock)
            {
                if(rightmostInImage)
                {
                    //color=imageData[pos+width]; // Already set
                    //component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+1)+1]=component;
                }
                else
                {
                    color=imageData[pos+width+1];
                    component=getFBWComponent(getFRed(color),getFGreen(color),getFBlue(color));
                    blockImageSection[blockImageSectionPos+(CUDA_BLOCK_SIDE_LENGTH+1)+1]=component;
                }
            }
        }
    }

    __syncthreads();

    float accXValueSum=0.0f;
    float accYValueSum=0.0f;

    // Right/bottom

    // X accumulator
    accXValueSum+=blockImageSection[(relativeY+1)*(CUDA_BLOCK_SIDE_LENGTH+1)+relativeX+1]*(-1.0f);

    // Right

    // Y accumulator
    accYValueSum+=blockImageSection[(relativeY)*(CUDA_BLOCK_SIDE_LENGTH+1)+relativeX+1]*(1.0f);

    // Bottom

    // Y accumulator
    accYValueSum+=blockImageSection[(relativeY+1)*(CUDA_BLOCK_SIDE_LENGTH+1)+relativeX]*(-1.0f);

    // This pixel:
    // X accumulator
    accXValueSum+=blockImageSection[(relativeY)*(CUDA_BLOCK_SIDE_LENGTH+1)+relativeX]*(1.0f);

    // Final result

    component=amplifier*sqrt(accXValueSum*accXValueSum+accYValueSum*accYValueSum);
    uint8_t componentOut=(uint8_t)round(component*255.0);
    imageDataOut[pos]=getColor(255,componentOut,componentOut,componentOut);
}

#define device_cudaGaussianBlur_imageDataPos(x) (4*(x)+channel)

__global__ void device_cudaGaussianBlur(uint8_t *imageData,int width,int height,int filterSize,float deviation,uint8_t *imageDataOut)
{
    const int blockImageSectionSingleChannelSizeSqrt=CUDA_3_CHANNEL_MAX_FILTER_SIZE*2+CUDA_GAUSSIAN_BLUR_BLOCK_SIDE_LENGTH;
    __shared__ float blockImageSection[blockImageSectionSingleChannelSizeSqrt*blockImageSectionSingleChannelSizeSqrt];

    int filterSizeInPixels=2*filterSize+1;
    int channel=blockIdx.z; // Only needed if on big-endian machine: add 1 for the alpha channel (to simplify memory ops)
    int blockHorId=blockIdx.x;
    int blockVerId=blockIdx.y;
    int relativeX=threadIdx.x;
    int x=blockHorId*blockDim.x+relativeX;
    int relativeY=threadIdx.y;
    int y=blockVerId*blockDim.y+relativeY;

    int rowSize=2*filterSize+blockDim.x;
    int columnSize=2*filterSize+blockDim.y;
    int topOffsetRows=filterSize;

    if(x>=width||y>=height)
        return;

    size_t pixelId=y*width+x;
    size_t pos=sizeof(uint32_t)*pixelId+channel;
    uint8_t component=imageData[pos];
    float color=colorComponentToF(component);

    // Copy pixels

    // Copy this pixel from source image

    int posInBlockImageSectionData=(topOffsetRows+relativeY)*rowSize+filterSize+relativeX;
    blockImageSection[posInBlockImageSectionData]=color;

    // Copy extra pixels, if needed

    int relativeLeftExtra=-filterSize+relativeX;
    int relativeRightExtra=relativeX+filterSize;
    int relativeTopExtra=-filterSize+relativeY;
    int relativeBottomExtra=relativeY+filterSize;

    int effectiveBlockWidth=blockHorId==gridDim.x-1?(width-blockHorId*blockDim.x):blockDim.x;
    int effectiveBlockHeight=blockVerId==gridDim.y-1?(height-blockVerId*blockDim.y):blockDim.y;

    bool copyExtraFromLeft=false;
    bool leftOverflow=false;
    bool copyExtraFromRight=false;
    bool rightOverflow=false;
    int absLeftExtra;
    int absRightExtra;

    if(copyExtraFromLeft=(relativeLeftExtra<0))
    {
        absLeftExtra=x-filterSize;
        if(leftOverflow=(absLeftExtra<0))
            blockImageSection[posInBlockImageSectionData-filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(y*width)]);
        else
            blockImageSection[posInBlockImageSectionData-filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(y*width+absLeftExtra)]);
    }
    if(copyExtraFromRight=(relativeRightExtra>=effectiveBlockWidth))
    {
        absRightExtra=x+filterSize;
        if(rightOverflow=(absRightExtra>=width))
            blockImageSection[posInBlockImageSectionData+filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(y*width+width-1)]);
        else
            blockImageSection[posInBlockImageSectionData+filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(y*width+absRightExtra)]);
    }
    if(relativeTopExtra<0)
    {
        int absTopExtra=y-filterSize;
        if(absTopExtra<0)
        {
            blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos((-absTopExtra)*width+x)]);
            if(copyExtraFromLeft)
            {
                if(leftOverflow)
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)-filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos((-absTopExtra)*width)]);
                else
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)-filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos((-absTopExtra)*width+absLeftExtra)]);
            }
            if(copyExtraFromRight)
            {
                if(rightOverflow)
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)+filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos((-absTopExtra)*width+width-1)]);
                else
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)+filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos((-absTopExtra)*width+absRightExtra)]);
            }
        }
        else
        {
            blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(absTopExtra*width+x)]);
            if(copyExtraFromLeft)
            {
                if(leftOverflow)
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)-filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(absTopExtra*width)]);
                else
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)-filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(absTopExtra*width+absLeftExtra)]);
            }
            if(copyExtraFromRight)
            {
                if(rightOverflow)
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)+filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(absTopExtra*width+width-1)]);
                else
                    blockImageSection[posInBlockImageSectionData-(filterSize*rowSize)+filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(absTopExtra*width+absRightExtra)]);
            }
        }
    }
    if(relativeBottomExtra>=effectiveBlockHeight)
    {
        int absBottomExtra=y+filterSize;
        if(absBottomExtra>=height)
        {
            blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos((height-(absBottomExtra-height+1))*width+x)]);
            if(copyExtraFromLeft)
            {
                if(leftOverflow)
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)-filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos((height-(absBottomExtra-height+1))*width)]);
                else
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)-filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos((height-(absBottomExtra-height+1))*width+absLeftExtra)]);
            }
            if(copyExtraFromRight)
            {
                if(rightOverflow)
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)+filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos((height-(absBottomExtra-height+1))*width+width-1)]);
                else
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)+filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos((height-(absBottomExtra-height+1))*width+absRightExtra)]);
            }
        }
        else
        {
            blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(absBottomExtra*width+x)]);
            if(copyExtraFromLeft)
            {
                if(leftOverflow)
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)-filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(absBottomExtra*width)]);
                else
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)-filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(absBottomExtra*width+absLeftExtra)]);
            }
            if(copyExtraFromRight)
            {
                if(rightOverflow)
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)+filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(absBottomExtra*width+width-1)]);
                else
                    blockImageSection[posInBlockImageSectionData+(filterSize*rowSize)+filterSize]=colorComponentToF(imageData[device_cudaGaussianBlur_imageDataPos(absBottomExtra*width+absRightExtra)]);
            }
        }
    }

    float preFactor=(1.0f/(2.0f*M_PI_F*deviation*deviation));
    float preFactor2=(2.0f*deviation*deviation);
    float filterFactors[CUDA_3_CHANNEL_MAX_FILTER_SIZE*CUDA_3_CHANNEL_MAX_FILTER_SIZE];
    for(int filterY=0;filterY<filterSizeInPixels;filterY++)
    {
        int offset=filterY*filterSizeInPixels;
        for(int filterX=0;filterX<filterSizeInPixels;filterX++)
        {
            int n1=filterX+1-filterSize-1;
            int n2=filterY+1-filterSize-1;
            float factor=preFactor*exp(-(((float)(n1*n1+n2*n2))/preFactor2));
            filterFactors[offset+filterX]=factor;
        }
    }

    __syncthreads();

    int relativeXWithBorderPixels=relativeX+filterSize;
    int relativeYWithBorderPixels=relativeY+filterSize;
    float pixelValueSum=0.0f;

    for(int yOfFilter=0;yOfFilter<filterSizeInPixels;yOfFilter++)
    {
        int yWithFilter=-filterSize+relativeYWithBorderPixels+yOfFilter;
         // Use symmetry to compensate for missing pixels (in order to avoid dark borders)
        if(yWithFilter<0)
            yWithFilter=(relativeYWithBorderPixels+filterSize)-yOfFilter;
        else if(yWithFilter>=columnSize)
            yWithFilter=(relativeYWithBorderPixels-filterSize)+(filterSizeInPixels-1-yOfFilter);
        for(int xOfFilter=0;xOfFilter<filterSizeInPixels;xOfFilter++)
        {
            int xWithFilter=-filterSize+relativeXWithBorderPixels+xOfFilter;
            // Use symmetry to compensate for missing pixels (in order to avoid dark borders)
            if(xWithFilter<0)
                xWithFilter=(relativeXWithBorderPixels+filterSize)-xOfFilter;
            else if(xWithFilter>=rowSize)
                xWithFilter=(relativeXWithBorderPixels-filterSize)+(filterSizeInPixels-1-xOfFilter);

            float factor=filterFactors[yOfFilter*filterSizeInPixels+xOfFilter];
            pixelValueSum+=blockImageSection[yWithFilter*rowSize+xWithFilter]*factor;
        }
    }

    imageDataOut[pos]=colorFToComponent(pixelValueSum);
    if(channel==0)
        imageDataOut[pos+3]=0xff;
}

#undef device_cudaGaussianBlur_imageDataPos

uint32_t *cudaGetBWImage(uint32_t *imageData,int width,int height)
{
    int blocksPerLine=ceil(floatDiv(width,CUDA_THREADS_PER_BLOCK));
    int totalNumBlocks=blocksPerLine*height; // Each block is responsible for a single line

    if(totalNumBlocks>65535)
        return 0;

    uint32_t *device_origImageData_in;
    uint32_t *device_newImageData_out;
    size_t imageSize=width*height*sizeof(uint32_t);
    hipMalloc(&device_origImageData_in,imageSize);
    hipMalloc(&device_newImageData_out,imageSize);

    hipMemcpy(device_origImageData_in,imageData,imageSize,hipMemcpyHostToDevice);

    dim3 blocks(blocksPerLine,height,1);

    device_getBWImage<<<blocks,CUDA_THREADS_PER_BLOCK>>>(device_origImageData_in,width,device_newImageData_out);

    uint32_t *newImageData=(uint32_t*)malloc(imageSize);
    hipMemcpy(newImageData,device_newImageData_out,imageSize,hipMemcpyDeviceToHost);

    hipFree(device_newImageData_out);
    hipFree(device_origImageData_in);

    return newImageData;
}

uint32_t *cudaCannyEdgeDetect(uint32_t *imageData,int width,int height,float deviation,float highTreshold,float lowTreshold)
{
    // Do not first convert to BW, as that would force us to copy memory back and forth

    // Do not use CUDA_THREADS_PER_BLOCK here!

    int horBlockCount=ceil(floatDiv(width,CUDA_BLOCK_SIDE_LENGTH));
    int verBlockCount=ceil(floatDiv(height,CUDA_BLOCK_SIDE_LENGTH));
    int totalNumBlocks=horBlockCount*verBlockCount;

    if(totalNumBlocks>65535)
        return 0;

    uint32_t *device_origImageData_in;
    uint32_t *device_newImageData_out;
    float *device_gaussianBwArray;
    float *device_gradientArray;
    float *device_gradientAtan2Array;
    uint8_t *device_preResultArray;
    size_t imageArea=width*height;
    size_t imageSize=imageArea*sizeof(uint32_t);
    size_t imageSizeFloat=imageArea*sizeof(float);
    hipMalloc(&device_origImageData_in,imageSize);
    hipMalloc(&device_newImageData_out,imageSize);
    hipMalloc(&device_gaussianBwArray,imageSizeFloat);
    hipMalloc(&device_gradientArray,imageSizeFloat);
    hipMalloc(&device_gradientAtan2Array,imageSizeFloat);
    hipMalloc(&device_preResultArray,imageArea*sizeof(uint8_t));

    hipMemcpy(device_origImageData_in,imageData,imageSize,hipMemcpyHostToDevice);

    dim3 blocks(horBlockCount,verBlockCount,1);
    dim3 threadsPerBlock(CUDA_BLOCK_SIDE_LENGTH,CUDA_BLOCK_SIDE_LENGTH,1);

    int filterSize=floor(deviation*3.0f); // NVidia standard

    device_cannyEdgeDetect_stage1<<<blocks,threadsPerBlock>>>(device_origImageData_in,width,height,filterSize,deviation,device_gaussianBwArray);
    device_cannyEdgeDetect_stage2<<<blocks,threadsPerBlock>>>(width,height,filterSize,device_gaussianBwArray,device_gradientArray,device_gradientAtan2Array);
    device_cannyEdgeDetect_stage3<<<blocks,threadsPerBlock>>>(width,height,highTreshold,lowTreshold,device_gradientArray,device_gradientAtan2Array,device_preResultArray);
    device_cannyEdgeDetect_stage4<<<blocks,threadsPerBlock>>>(width,height,device_preResultArray,device_newImageData_out);

    uint32_t *newImageData=(uint32_t*)malloc(imageSize);
    hipMemcpy(newImageData,device_newImageData_out,imageSize,hipMemcpyDeviceToHost);

    hipFree(device_preResultArray);
    hipFree(device_gradientAtan2Array);
    hipFree(device_gradientArray);
    hipFree(device_gaussianBwArray);
    hipFree(device_newImageData_out);
    hipFree(device_origImageData_in);

    return newImageData;
}

uint32_t *cudaSobelEdgeDetect(uint32_t *imageData,int width,int height,float amplifier)
{
    // Do not first convert to BW, as that would force us to copy memory back and forth

    // Do not use CUDA_THREADS_PER_BLOCK here!

    int horBlockCount=ceil(floatDiv(width,/*!!!*/CUDA_BLOCK_SIDE_LENGTH));
    int verBlockCount=ceil(floatDiv(height,/*!!!*/CUDA_BLOCK_SIDE_LENGTH));
    int totalNumBlocks=horBlockCount*verBlockCount;

    if(totalNumBlocks>65535)
        return 0;

    uint32_t *device_origImageData_in;
    uint32_t *device_newImageData_out;
    size_t imageSize=width*height*sizeof(uint32_t);
    hipMalloc(&device_origImageData_in,imageSize);
    hipMalloc(&device_newImageData_out,imageSize);

    hipMemcpy(device_origImageData_in,imageData,imageSize,hipMemcpyHostToDevice);

    dim3 blocks(horBlockCount,verBlockCount,1);
    dim3 threadsPerBlock(CUDA_BLOCK_SIDE_LENGTH,CUDA_BLOCK_SIDE_LENGTH,1);

    device_sobelEdgeDetect<<<blocks,threadsPerBlock>>>(device_origImageData_in,width,height,amplifier,device_newImageData_out);

    uint32_t *newImageData=(uint32_t*)malloc(imageSize);
    hipMemcpy(newImageData,device_newImageData_out,imageSize,hipMemcpyDeviceToHost);

    hipFree(device_newImageData_out);
    hipFree(device_origImageData_in);

    return newImageData;
}

uint32_t *cudaRobertsEdgeDetect(uint32_t *imageData,int width,int height,float amplifier)
{
    // Do not first convert to BW, as that would force us to copy memory back and forth

    // Do not use CUDA_THREADS_PER_BLOCK here!

    int horBlockCount=ceil(floatDiv(width,/*!!!*/CUDA_BLOCK_SIDE_LENGTH));
    int verBlockCount=ceil(floatDiv(height,/*!!!*/CUDA_BLOCK_SIDE_LENGTH));
    int totalNumBlocks=horBlockCount*verBlockCount;

    if(totalNumBlocks>65535)
        return 0;

    uint32_t *device_origImageData_in;
    uint32_t *device_newImageData_out;
    size_t imageSize=width*height*sizeof(uint32_t);
    hipMalloc(&device_origImageData_in,imageSize);
    hipMalloc(&device_newImageData_out,imageSize);

    hipMemcpy(device_origImageData_in,imageData,imageSize,hipMemcpyHostToDevice);

    dim3 blocks(horBlockCount,verBlockCount,1);
    dim3 threadsPerBlock(CUDA_BLOCK_SIDE_LENGTH,CUDA_BLOCK_SIDE_LENGTH,1);

    device_robertsEdgeDetect<<<blocks,threadsPerBlock>>>(device_origImageData_in,width,height,amplifier,device_newImageData_out);

    uint32_t *newImageData=(uint32_t*)malloc(imageSize);
    hipMemcpy(newImageData,device_newImageData_out,imageSize,hipMemcpyDeviceToHost);

    hipFree(device_newImageData_out);
    hipFree(device_origImageData_in);

    return newImageData;
}

uint32_t *cuda3ChannelGaussianBlur(uint32_t *imageData,int width,int height,int filterSize,float deviation)
{
    // Do not first convert to BW, as that would force us to copy memory back and forth

    // Do not use CUDA_THREADS_PER_BLOCK here!

    int horBlockCount=ceil(floatDiv(width,/*!!!*/CUDA_GAUSSIAN_BLUR_BLOCK_SIDE_LENGTH));
    int verBlockCount=ceil(floatDiv(height,/*!!!*/CUDA_GAUSSIAN_BLUR_BLOCK_SIDE_LENGTH));
    int totalNumBlocks=horBlockCount*verBlockCount;

    if(totalNumBlocks*3>65535||filterSize>CUDA_3_CHANNEL_MAX_FILTER_SIZE)
        return 0;

    uint8_t *device_origImageData_in;
    uint8_t *device_newImageData_out;
    size_t imageArea=width*height;
    size_t imageSize=imageArea*sizeof(uint32_t);
    hipMalloc(&device_origImageData_in,imageSize);
    hipMalloc(&device_newImageData_out,imageSize);

    hipMemcpy(device_origImageData_in,imageData,imageSize,hipMemcpyHostToDevice);

    dim3 blocks(horBlockCount,verBlockCount,3); // 3: R, G and B channel
    dim3 threadsPerBlock(CUDA_GAUSSIAN_BLUR_BLOCK_SIDE_LENGTH,CUDA_GAUSSIAN_BLUR_BLOCK_SIDE_LENGTH,1);

    device_cudaGaussianBlur<<<blocks,threadsPerBlock>>>(device_origImageData_in,width,height,filterSize,deviation,device_newImageData_out);

    uint32_t *newImageData=(uint32_t*)malloc(imageSize);
    hipMemcpy(newImageData,device_newImageData_out,imageSize,hipMemcpyDeviceToHost);

    // Alpha channel filled in device_cudaGaussianBlur
    //for(size_t i=0;i<imageArea;i++) // Fill alpha channel
    //    newImageData[i]|=0xff000000;

    hipFree(device_newImageData_out);
    hipFree(device_origImageData_in);

    return newImageData;
}
